/* Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

// reference: https://github.com/NVIDIA/TensorRT-LLM/blob/release/0.14/cpp/tensorrt_llm/kernels/customAllReduceKernels.h

#include <c10/cuda/CUDAStream.h>

#include <cassert>

#include "trt_reduce_internal.cuh"
#include "utils.h"

using namespace trt_llm;

using fptr_t = int64_t;
using IPC_KEY = std::array<uint8_t, sizeof(hipIpcMemHandle_t)>;

class AllReduceMeta {
 public:
  AllReduceMeta(int64_t rank_id, int64_t world_size, torch::Tensor& rank_data, const std::vector<fptr_t>& buffers,
                const std::vector<fptr_t>& tmp_result_buffers, const std::vector<fptr_t>& barrier_in,
                const std::vector<fptr_t>& barrier_out) {
    this->rank_id = (int)rank_id;
    this->world_size = (int)world_size;
    this->barrier_in = barrier_in;
    this->barrier_out = barrier_out;
    this->tmp_result_buffers = tmp_result_buffers;

    this->rank_data_base = reinterpret_cast<RankData*>(rank_data.data_ptr());
    RankData data;
    for (int i = 0; i < world_size; i++) {
      data.ptrs[i] = (void*)buffers[i];
    }
    auto d_data = this->rank_data_base++;
    CHECK_CUDA_SUCCESS(hipMemcpy(d_data, &data, sizeof(RankData), hipMemcpyHostToDevice));
    this->buffers = d_data;
  }

  ~AllReduceMeta() {
    for (auto [_, ptr] : ipc_handles_) {
      CHECK_CUDA_SUCCESS(hipIpcCloseMemHandle(ptr));
    }
  }

 public:
  int world_size;
  int rank_id;
  std::vector<fptr_t> barrier_in;
  std::vector<fptr_t> barrier_out;
  std::vector<fptr_t> tmp_result_buffers;
  int barrier_flag = 1;
  RankData* buffers;
  RankData* rank_data_base;
  std::vector<void*> graph_unreg_buffers;
  std::map<IPC_KEY, char*> ipc_handles_;
};

// Get the number of bits for a given data type.
inline int get_bits(at::ScalarType dtype) {
  switch (dtype) {
    case at::ScalarType::Float:
      return 32;
    case at::ScalarType::Half:
    case at::ScalarType::BFloat16:
      return 16;
    default:
      assert(false && "Unsupported data type");
  }
}

// Check if customized all-reduce kernels can be applied.
inline bool CanApplyCustomAllReduce(int64_t num_elements, at::ScalarType dtype) {
  // The customized all-reduce kernel has the following requirement(s).
  return num_elements % (16 / ((get_bits(dtype) + 7) / 8)) == 0;
}

fptr_t init_custom_ar(int64_t rank_id, int64_t world_size, torch::Tensor& rank_data, const std::vector<fptr_t>& buffers,
                      const std::vector<fptr_t>& tmp_result_buffers, const std::vector<fptr_t>& barrier_in,
                      const std::vector<fptr_t>& barrier_out) {
  auto m = new AllReduceMeta(rank_id, world_size, rank_data, buffers, tmp_result_buffers, barrier_in, barrier_out);
  return (fptr_t)m;
}

void dispose(fptr_t _fa) {
  auto fa = reinterpret_cast<AllReduceMeta*>(_fa);
  delete fa;
}

std::tuple<std::vector<int64_t>, std::vector<int64_t>> get_graph_buffer_ipc_meta(fptr_t _fa) {
  AllReduceMeta* m = reinterpret_cast<AllReduceMeta*>(_fa);
  auto num_buffers = m->graph_unreg_buffers.size();
  auto handle_sz = sizeof(hipIpcMemHandle_t);
  std::string handles(handle_sz * num_buffers, static_cast<char>(0));
  std::vector<int64_t> offsets(num_buffers);
  for (int i = 0; i < num_buffers; i++) {
    auto ptr = m->graph_unreg_buffers[i];
    void* base_ptr;
    // note: must share the base address of each allocation, or we get wrong
    // address
    if (hipPointerGetAttribute(&base_ptr, HIP_POINTER_ATTRIBUTE_RANGE_START_ADDR, (hipDeviceptr_t)ptr) != hipSuccess) {
      assert(false && "failed to get pointer attr");
    }

    CHECK_CUDA_SUCCESS(hipIpcGetMemHandle((hipIpcMemHandle_t*)&handles[i * handle_sz], base_ptr));
    offsets[i] = ((char*)ptr) - ((char*)base_ptr);
  }
  std::vector<int64_t> bytes(handles.begin(), handles.end());
  return std::make_pair(bytes, offsets);
}

char* open_ipc_handle(AllReduceMeta* meta, const void* ipc_handle) {
  auto [it, new_handle] = meta->ipc_handles_.insert({*((IPC_KEY*)ipc_handle), nullptr});
  if (new_handle) {
    char* ipc_ptr;
    CHECK_CUDA_SUCCESS(hipIpcOpenMemHandle((void**)&ipc_ptr, *((const hipIpcMemHandle_t*)ipc_handle),
                                            hipIpcMemLazyEnablePeerAccess));
    it->second = ipc_ptr;
  }
  return it->second;
}

// Note: when registering graph buffers, we intentionally choose to not
// deduplicate the addresses. That means if the allocator reuses some
// addresses, they will be registered again. This is to account for the remote
// possibility of different allocation patterns between ranks. For example,
// rank 1 may get the same input address for the second allreduce, but rank 2
// got a different address. IPC handles have internal reference counting
// mechanism so overhead should be small.
void register_graph_buffers(fptr_t _fa, const std::vector<std::vector<int64_t>>& handles,
                            const std::vector<std::vector<int64_t>>& offsets) {
  AllReduceMeta* m = reinterpret_cast<AllReduceMeta*>(_fa);
  std::vector<std::string> handle_bytes;
  handle_bytes.reserve(handles.size());
  for (int i = 0; i < handles.size(); i++) {
    handle_bytes.emplace_back(handles[i].begin(), handles[i].end());
  }
  auto num_buffers = m->graph_unreg_buffers.size();
  std::vector<RankData> rank_data(num_buffers);
  for (int i = 0; i < num_buffers; i++) {
    auto self_ptr = m->graph_unreg_buffers[i];
    auto& rd = rank_data[i];
    for (int j = 0; j < m->world_size; j++) {
      if (j != m->rank_id) {
        char* handle = open_ipc_handle(m, &handle_bytes[j][i * sizeof(hipIpcMemHandle_t)]);
        handle += offsets[j][i];
        rd.ptrs[j] = handle;
      } else {
        rd.ptrs[j] = self_ptr;
      }
    }
  }
  CHECK_CUDA_SUCCESS(
      hipMemcpy(m->rank_data_base, rank_data.data(), sizeof(RankData) * num_buffers, hipMemcpyHostToDevice));
  m->rank_data_base += num_buffers;
  m->graph_unreg_buffers.clear();
}

void all_reduce(fptr_t _fa, torch::Tensor& inp, torch::Tensor& out) {
  AllReduceMeta* m = reinterpret_cast<AllReduceMeta*>(_fa);
  auto stream = c10::cuda::getCurrentCUDAStream().stream();
  auto num_elements = inp.numel();
  auto dtype = inp.scalar_type();
  AllReduceStrategyType strategy = SelectImplementation(num_elements * ((get_bits(dtype) + 7) / 8), m->world_size);

  // should be gurantee in python code
  assert(strategy == AllReduceStrategyType::ONESHOT || strategy == AllReduceStrategyType::TWOSHOT);
  assert(CanApplyCustomAllReduce(num_elements, dtype));

  // Initialize the all-reduce kernel arguments.
  int world_size = m->world_size;

  AllReduceParams params;
  params.ranks_per_node = world_size;
  params.rank = m->rank_id;
  params.local_rank = m->rank_id;
  params.local_input_buffer_ptr = inp.data_ptr();
  params.local_output_buffer_ptr = out.data_ptr();
  params.elts_total = inp.numel();
  params.elts_size = inp.element_size();
  params.barrier_flag = ++(m->barrier_flag);

  hipStreamCaptureStatus status;
  CHECK_CUDA_SUCCESS(hipStreamIsCapturing(stream, &status));
  params.is_capturing = (status == hipStreamCaptureStatusActive);
  if (params.is_capturing) {
    params.peer_comm_buffer_ptrs = m->rank_data_base + m->graph_unreg_buffers.size();
    m->graph_unreg_buffers.push_back(params.local_input_buffer_ptr);
  } else {
    params.peer_comm_buffer_ptrs = m->buffers;
  }

  for (int i = 0; i < world_size; ++i) {
    params.tmp_result_buffers[i] = reinterpret_cast<uint32_t*>(m->tmp_result_buffers[i]);
  }
  for (int i = 0; i < world_size; ++i) {
    params.peer_barrier_ptrs_in[i] = reinterpret_cast<uint32_t*>(m->barrier_in[i]);
  }
  for (int i = 0; i < world_size; ++i) {
    params.peer_barrier_ptrs_out[i] = reinterpret_cast<uint32_t*>(m->barrier_out[i]);
  }

  auto data_type = out.scalar_type();
  trtCustomAllReduce(params, data_type, strategy, stream);
}
