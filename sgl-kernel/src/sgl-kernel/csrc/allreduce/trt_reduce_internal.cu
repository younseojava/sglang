#include "hip/hip_runtime.h"
/* Copyright 2025 SGLang Team. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

// reference:
// https://github.com/NVIDIA/TensorRT-LLM/blob/release/0.14/cpp/tensorrt_llm/kernels/customAllReduceKernels.cu
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

#include <cassert>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <tuple>

#include "trt_reduce_internal.cuh"
#include "utils.h"

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void st_flag_release(uint32_t const& flag, uint32_t* flag_addr) {
  asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t ld_flag_acquire(uint32_t* flag_addr) {
  uint32_t flag;
  asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
  return flag;
}

static inline __device__ void st_flag_volatile(uint32_t const& flag, uint32_t* flag_addr) {
  asm volatile("st.volatile.global.u32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
}

static inline __device__ uint32_t ld_flag_volatile(uint32_t* flag_addr) {
  uint32_t flag;
  asm volatile("ld.volatile.global.u32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
  return flag;
}

namespace trt_llm {
////////////////////////////////////////////////////////////////////////////////////////////////////

// Type Converter that packs data format to 128 bits data type
//
using PackedFloat = union {
  int4 packed;
  float unpacked[4];
};

using PackedHalf = union {
  int4 packed;
  half2 unpacked[4];
};

template <typename T>
struct PackedOn16Bytes {};

template <>
struct PackedOn16Bytes<float> {
  using Type = PackedFloat;
};

template <>
struct PackedOn16Bytes<half> {
  using Type = PackedHalf;
};

#if (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
using PackedBFloat16 = union {
  int4 packed;
  __hip_bfloat162 unpacked[4];
};

template <>
struct PackedOn16Bytes<__hip_bfloat16> {
  using Type = PackedBFloat16;
};
#endif

// add two 128b data
template <typename T>
inline __device__ int4 add128b(T& a, T& b) {
  T c;
  c.unpacked[0] = a.unpacked[0] + b.unpacked[0];
  c.unpacked[1] = a.unpacked[1] + b.unpacked[1];
  c.unpacked[2] = a.unpacked[2] + b.unpacked[2];
  c.unpacked[3] = a.unpacked[3] + b.unpacked[3];
  return c.packed;
}

__inline__ __device__ void multi_gpu_barrier(uint32_t** signals, uint32_t const flag, size_t const local_rank,
                                             size_t const world_size, int const tidx, int const bidx) {
  // After this function, at least one block in each GPU has reached the barrier
  if (tidx < world_size) {
    // we can think of signals having the shape [world_size, world_size]
    // Dimension 0 is the "listening" dimension, dimension 1 is "emitting" dimension

    // Block 0 broadcasts its flag (local_rank on emitting dimension) to all receivers
    size_t offset = (flag % 2) ? world_size : 0;

    if (bidx == 0) {
      st_flag_release(flag, signals[tidx] + offset + local_rank);
    }

    // All blocks check that corresponding block 0 on other GPUs have set the flag
    // No deadlock because block #0 is always the first block started
    uint32_t* peer_barrier_d = signals[local_rank] + offset + tidx;
    while (ld_flag_acquire(peer_barrier_d) != flag) {
    }
  }

  __syncthreads();
}

template <bool start, bool need_fence = false>
__inline__ __device__ void block_barrier(uint32_t** signals, uint32_t const flag, size_t const local_rank,
                                         size_t const world_size, int const tidx, int const bidx, int const grid_size) {
  if constexpr (!start) {
    __syncthreads();
  }
  // After this function, the block of id == bidx of each GPU has reached the barrier
  if (tidx < world_size) {
    // we can think of signals having the shape [world_size, 2, num_blocks, world_size]
    // (+ an offset on dim 2 to account for flags used in multi_gpu_barrier)
    // Dimension 0 is the "listening" dimension, dimension 3 is "emitting" dimension

    // Block broadcast its flag (local_rank on emitting dimension) to all receivers
    uint32_t flag_block_offset = world_size + bidx * world_size;

    flag_block_offset += (grid_size + 1) * world_size * (flag % 2);

    uint32_t* peer_barrier_d = signals[local_rank] + flag_block_offset + tidx;
    // Blocks check that corresponding blocks on other GPUs have also set the flag
    if constexpr (need_fence) {
      st_flag_release(flag, signals[tidx] + flag_block_offset + local_rank);
      while (ld_flag_acquire(peer_barrier_d) != flag) {
      }
    } else {
      st_flag_volatile(flag, signals[tidx] + flag_block_offset + local_rank);
      while (ld_flag_volatile(peer_barrier_d) != flag) {
      }
    }
  }

  __syncthreads();
}

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT = true>
static __global__ void __launch_bounds__(512, 1) oneShotAllReduceKernel(AllReduceParams params) {
  // Suppose that two GPUs participate in the AR exchange, and we start four blocks.
  // The message is partitioned into chunks as detailed below:
  //               message
  //       |-------------------|
  // GPU 0 | B0 | B1 | B2 | B3 |
  // GPU 1 | B0 | B1 | B2 | B3 |
  //
  // Here the step-by-step behavior of one block:
  // 1. B0 copies the chunk it  is responsible for, from local_input to shareable buffer
  // 2. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier)
  // 3. B0 on GPU 0 pull and sum the chunk from GPU 1, writes the result to local_output
  //
  // With COPY_INPUT == false, skip step 1. and use gpu_barrier instead of block barrier during step 2.
  // We only to know if the other GPU as arrived at the AR kernel, that would mean that data is ready
  //
  // With PUSH_MODE, we consider that the shared buffer is of size:
  // params.peer_comm_buffer_ptrs: [world_size, world_size, message_size]
  //
  // Here the step-by-step behavior of one block:
  // 1. B0 push the chunk is it responsible for into all other GPUs:
  //    params.peer_comm_buffer_ptrs[:, local_gpu, B0 slice]
  // 2. block sync so the block is shared by other GPUs
  // 3. Reduce along second dimension params.peer_comm_buffer_ptrs[local_gpu, :, B0 slice]

  int const bidx = blockIdx.x;
  int const tidx = threadIdx.x;
  int const grid_size = gridDim.x;

  // The number of elements packed into one for comms
  static constexpr int NUM_ELTS = 16 / sizeof(T);

  // Packed data type for comms
  using PackedStruct = typename PackedOn16Bytes<T>::Type;

  // The source pointers. Distributed round-robin for the different warps.
  auto peer_comm_buffer_ptrs = params.peer_comm_buffer_ptrs->ptrs;
  T* local_shared_buffer = reinterpret_cast<T*>(peer_comm_buffer_ptrs[params.local_rank]);
  // Start and end offsets of the thread
  size_t chunk_start = bidx * params.elts_per_block + tidx * NUM_ELTS;
  size_t chunk_end = std::min((bidx + 1) * params.elts_per_block, params.elts_per_rank);

  if constexpr (COPY_INPUT) {
    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    // Copy from local buffer to shareable buffer
    for (size_t iter_offset = chunk_start; iter_offset < chunk_end; iter_offset += blockDim.x * NUM_ELTS) {
      *reinterpret_cast<int4*>(&local_shared_buffer[iter_offset]) =
          *reinterpret_cast<int4 const*>(&local_input_buffer[iter_offset]);
    }
  }
  // wait for equivalent blocks of other GPUs to have copied data to their shareable buffer
  block_barrier<true>(params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx,
                      grid_size);

  // Each block accumulates the values from the different GPUs on the same node.
  for (size_t iter_offset = chunk_start; iter_offset < chunk_end; iter_offset += blockDim.x * NUM_ELTS) {
    // Iterate over the different ranks/devices on the node to load the values.
    PackedStruct vals[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
      vals[ii].packed = *reinterpret_cast<int4 const*>(&((T*)peer_comm_buffer_ptrs[ii])[iter_offset]);
    }

    // Sum the values from the different ranks.
    PackedStruct sums;
    sums.packed = {0, 0, 0, 0};
#pragma unroll
    for (int rank = 0; rank < RANKS_PER_NODE; ++rank) {
      // Always reduce from rank 0 to ensure stable reduce order.
      sums.packed = add128b(sums, vals[rank]);
    }

    // Store to the destination buffer.
    *reinterpret_cast<int4*>(&reinterpret_cast<T*>(params.local_output_buffer_ptr)[iter_offset]) = sums.packed;
  }
}

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT = true>
static __global__ void __launch_bounds__(512, 1) twoShotAllReduceKernel(AllReduceParams params) {
  // Suppose that two GPUs participate in the AR exchange, and we start two blocks.
  // The message is partitioned into chunks as detailed below:
  //               message
  //       |-------------------|
  //       |--GPU 0--|--GPU 1--| (GPU responsibility parts)
  // GPU 0 | B0 | B1 | B0 | B1 |
  // GPU 1 | B0 | B1 | B0 | B1 |
  //
  // Here the step-by-step behavior of one block:
  // 1. B0 copies all chunks is it responsible for, from local_input to shareable buffer
  // 2. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier #0)
  // 3. B0 on GPU 0 gather and sum the B0 chunks from GPU 1, that are in the GPU 0 responsibility
  //    part (the first half of the message, see GPU responsibility row above)
  // 3bis. Likewise, B0 on GPU 1 copies and sum the chunks for GPU 0,
  //       where GPU 1 is responsible: the second half of the message.
  // 4. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier #1)
  // 5. B0 writes result to local_output. It gathers each chunk from its responsible GPU.
  //    For example, here it reads the first chunk from GPU 0 and second chunk from GPU 1.
  //
  // With COPY_INPUT == false, skip step 1. and use gpu_barrier instead of block barrier during step 2.
  // We only to know if the other GPU as arrived at the AR kernel, that would mean that data is ready
  // to be read.
  //
  // Note that compared to one-shot, one block (CTA) writes multiple input chunks and write multiple output chunks.
  // However, it's only responsible for the summation of a single chunk.
  //
  // With PUSH_MODE, we consider that the shared buffer is of size:
  // params.peer_comm_buffer_ptrs: [world_size, world_size, message_size / world_size]
  //
  // Here the step-by-step behavior of one block:
  // 1. B0 push the chunks is it responsible for into the corresponding GPUs:
  //    params.peer_comm_buffer_ptrs[target_gpu, local_gpu, current B0 slice]
  // 2. block sync so the blocks have been shared by other GPUs
  // 3. Reduce along second dimension params.peer_comm_buffer_ptrs[local_gpu, :, B0 slice]
  // 4. block barrier (corresponding blocks have finished reduction)
  // 5. pull and write on local buffer, by reading params.peer_comm_buffer_ptrs[:, 0, B0 slice] (reduction result is
  //    written at index 0 of 2nd dim)

  int const bidx = blockIdx.x;
  int const tidx = threadIdx.x;
  int const grid_size = gridDim.x;

  // The number of elements packed into one for comms
  static constexpr int PACKED_ELTS = 16 / sizeof(T);
  using PackedType = typename PackedOn16Bytes<T>::Type;

  T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
  auto peer_comm_buffer_ptrs = params.peer_comm_buffer_ptrs->ptrs;
  T* local_shared_buffer = reinterpret_cast<T*>(peer_comm_buffer_ptrs[params.local_rank]);
  T* local_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);

  size_t const chunk_start = bidx * params.elts_per_block + tidx * PACKED_ELTS;
  size_t const chunk_end = min(chunk_start + params.elts_per_block, params.elts_per_rank);

  T* buffers[RANKS_PER_NODE];
  T* buffers_unorder[RANKS_PER_NODE];
  int ranks[RANKS_PER_NODE];
#pragma unroll
  for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
    // A mapping of the ranks to scatter reads as much as possible
    int rank = (params.local_rank + ii) % RANKS_PER_NODE;
    ranks[ii] = rank;
    buffers[ii] = reinterpret_cast<T*>(peer_comm_buffer_ptrs[rank]);
    buffers_unorder[ii] = reinterpret_cast<T*>(peer_comm_buffer_ptrs[ii]);
  }

#if (defined(__CUDACC_VER_MAJOR__) && (__CUDACC_VER_MAJOR__ >= 12))
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
  cudaGridDependencySynchronize();
#endif
#endif

  if constexpr (COPY_INPUT) {
    // Copy all blocks from local buffer to shareable buffer
    for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS) {
#pragma unroll
      for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
        size_t offset_rank = ranks[ii] * params.elts_per_rank + local_offset;
        if (offset_rank >= params.elts_total) {
          continue;
        }
        *reinterpret_cast<int4*>(&local_shared_buffer[offset_rank]) =
            *reinterpret_cast<int4 const*>(&local_input_buffer[offset_rank]);
      }
    }
  }
  block_barrier<true>(params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx,
                      grid_size);

  // Each block accumulates the values from the different GPUs on the same node.
  for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS) {
    size_t const responsible_block_offset = local_offset + params.rank_offset;

    // Iterate over the different ranks/devices on the node to load the values.
    PackedType vals[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
      vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers_unorder[ii][responsible_block_offset]);
    }

    // Sum the values from the different ranks.
    PackedType sums;
    sums.packed = {0, 0, 0, 0};
#pragma unroll
    for (int rank = 0; rank < RANKS_PER_NODE; ++rank) {
      // Always reduce from rank 0 to ensure stable reduce order.
      sums.packed = add128b(sums, vals[rank]);
    }

    // Store to the local buffer or tmp buffer
    if constexpr (COPY_INPUT) {
      *reinterpret_cast<int4*>(&local_shared_buffer[responsible_block_offset]) = sums.packed;
    } else {
      *reinterpret_cast<int4*>(&params.tmp_result_buffers[params.local_rank][responsible_block_offset]) = sums.packed;
    }
  }

  block_barrier<false, true>(params.peer_barrier_ptrs_out, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx,
                             bidx, grid_size);

  // Gather all needed elts from other intra-node ranks
  for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS) {
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii) {
      // use round-robin gathering from other ranks
      size_t offset_rank = ranks[ii] * params.elts_per_rank + local_offset;
      if (offset_rank >= params.elts_total) {
        continue;
      }
      if constexpr (COPY_INPUT) {
        *reinterpret_cast<int4*>(&local_output_buffer[offset_rank]) =
            *reinterpret_cast<int4*>(&buffers[ii][offset_rank]);
      } else {
        *reinterpret_cast<int4*>(&local_output_buffer[offset_rank]) =
            *reinterpret_cast<int4*>(&params.tmp_result_buffers[ranks[ii]][offset_rank]);
      }
    }
  }
#if (defined(__CUDACC_VER_MAJOR__) && (__CUDACC_VER_MAJOR__ >= 12))
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
  cudaTriggerProgrammaticLaunchCompletion();
#endif
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

inline int divUp(int a, int b) {
  return (a + b - 1) / b;
}

inline int roundUp(int a, int n) {
  return divUp(a, n) * n;
}

std::tuple<int, int> kernelLaunchConfig(AllReduceStrategyType algo, AllReduceParams& params, size_t elts_per_thread) {
  int blocks_per_grid = 1, threads_per_block = DEFAULT_BLOCK_SIZE;
  switch (algo) {
    case AllReduceStrategyType::ONESHOT: {
      assert(params.elts_total % elts_per_thread == 0);
      size_t const total_threads = roundUp(params.elts_total / elts_per_thread, WARP_SIZE);
      threads_per_block = std::min(DEFAULT_BLOCK_SIZE, total_threads);
      blocks_per_grid = std::min(static_cast<int>(MAX_ALL_REDUCE_BLOCKS), divUp(total_threads, threads_per_block));
      params.elts_per_block = roundUp(divUp(params.elts_total, blocks_per_grid), elts_per_thread);
      params.elts_per_rank = params.elts_total;
      break;
    }
    case AllReduceStrategyType::TWOSHOT: {
      assert(params.elts_total % (elts_per_thread * params.ranks_per_node) == 0);
      size_t const total_threads = roundUp(params.elts_total / (elts_per_thread * params.ranks_per_node), WARP_SIZE);

      /*
      threads_per_block = std::min(DEFAULT_BLOCK_SIZE, total_threads);
      blocks_per_grid = std::min(static_cast<size_t>(MAX_ALL_REDUCE_BLOCKS), divUp(total_threads, threads_per_block));
      */
      while (total_threads % blocks_per_grid != 0 || total_threads / blocks_per_grid > DEFAULT_BLOCK_SIZE) {
        blocks_per_grid += 1;
      }

      threads_per_block = total_threads / blocks_per_grid;

      // NOTE: need to adjust here
      if (blocks_per_grid > MAX_ALL_REDUCE_BLOCKS) {
        size_t iter_factor = 1;
        while (blocks_per_grid / iter_factor > MAX_ALL_REDUCE_BLOCKS || blocks_per_grid % iter_factor) {
          iter_factor += 1;
        }
        blocks_per_grid /= iter_factor;
      }
      params.elts_per_rank = params.elts_total / params.ranks_per_node;
      params.rank_offset = params.local_rank * params.elts_per_rank;
      params.elts_per_block = roundUp(divUp(params.elts_per_rank, blocks_per_grid), elts_per_thread);
      break;
    }
    default:
      assert(false && "Algorithm not supported here.");
  }

  return std::make_tuple(blocks_per_grid, threads_per_block);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT>
void dispatchARKernels(AllReduceStrategyType algo, AllReduceParams& param, int blocks_per_grid, int threads_per_block,
                       hipStream_t stream) {
  switch (algo) {
    case AllReduceStrategyType::ONESHOT: {
      oneShotAllReduceKernel<T, RANKS_PER_NODE, COPY_INPUT><<<blocks_per_grid, threads_per_block, 0, stream>>>(param);
      break;
    }
    case AllReduceStrategyType::TWOSHOT: {
      twoShotAllReduceKernel<T, RANKS_PER_NODE, COPY_INPUT><<<blocks_per_grid, threads_per_block, 0, stream>>>(param);
      break;
    }
  }
}

template <typename T, bool COPY_INPUT>
void dispatchARKernelsCopyInput(AllReduceStrategyType strat, AllReduceParams& param, hipStream_t stream) {
  size_t elts_per_thread = 16 / sizeof(T);
  auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(strat, param, elts_per_thread);
  switch (param.ranks_per_node) {
    case 2:
      dispatchARKernels<T, 2, COPY_INPUT>(strat, param, blocks_per_grid, threads_per_block, stream);
      break;
    case 4:
      dispatchARKernels<T, 4, COPY_INPUT>(strat, param, blocks_per_grid, threads_per_block, stream);
      break;
    case 6:
      dispatchARKernels<T, 6, COPY_INPUT>(strat, param, blocks_per_grid, threads_per_block, stream);
      break;
    case 8:
      dispatchARKernels<T, 8, COPY_INPUT>(strat, param, blocks_per_grid, threads_per_block, stream);
      break;
    default:
      break;
  }
}

template <typename T>
void invokeOneOrTwoShotAllReduceKernel(AllReduceParams& param, AllReduceStrategyType strat, hipStream_t stream) {
  if (param.is_capturing) {
    dispatchARKernelsCopyInput<T, false>(strat, param, stream);
  } else {
    dispatchARKernelsCopyInput<T, true>(strat, param, stream);
  }
  CHECK_CUDA_SUCCESS(hipGetLastError());
}

void trtCustomAllReduce(AllReduceParams& params, at::ScalarType data_type, AllReduceStrategyType strat,
                        hipStream_t stream) {
  if (params.elts_total == 0) {
    return;
  }

  switch (data_type) {
    case at::ScalarType::Float:
      invokeOneOrTwoShotAllReduceKernel<float>(params, strat, stream);
      break;
    case at::ScalarType::Half:
      invokeOneOrTwoShotAllReduceKernel<half>(params, strat, stream);
      break;
#if (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
    case at::ScalarType::BFloat16:
      invokeOneOrTwoShotAllReduceKernel<__hip_bfloat16>(params, strat, stream);
      break;
#endif
    default:
      assert(false && "Unsupported data type");
  }
}
}  // namespace trt_llm
