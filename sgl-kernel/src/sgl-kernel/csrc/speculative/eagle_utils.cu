#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025 by SGLang team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

// parent_list [bs, topk * (depth - 1) + 1)]
// selected_index [bs, draft_token_num - 1]
// verified_seq_len [bs]
// tree_mask [draft_token*(seq_len[0]+draft_token) | draft_token*(seq_len[1]+draft_token) | ..] =
// [sum(verified_seq_len)*draft_token+bs*draft_token*draft_token] positions [bs * draft_token] retrive_index [b,
// draft_token] retrive_next_token [b, draft_token] retrive_next_sibling [b, draft_token]
__global__ void build_tree_efficient(int64_t* parent_list, int64_t* selected_index, int32_t* verified_seq_len,
                                     bool* tree_mask, int64_t* positions, int64_t* retrive_index,
                                     int64_t* retrive_next_token, int64_t* retrive_next_sibling, int topk, int depth,
                                     int draft_token_num) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid >= draft_token_num) {
    return;
  }
  int seq_tree_idx = draft_token_num * draft_token_num * bid;
  for (int i = 0; i < bid; i++) {
    seq_tree_idx += verified_seq_len[i] * draft_token_num;
  }
  int seq_len = verified_seq_len[bid];
  int token_tree_idx = seq_tree_idx + (seq_len + draft_token_num) * tid + seq_len + 1;
  for (int i = 0; i < draft_token_num - 1; i++) {
    tree_mask[token_tree_idx + i] = false;
  }

  int position = 0;
  if (tid == 0) {
    positions[bid * draft_token_num] = seq_len;

    int retrive_index_offset = bid * draft_token_num;
    for (int i = draft_token_num - 1; i > 0; --i) {
      int current_token_idx = retrive_index_offset + i;
      retrive_index[bid * draft_token_num + i] = current_token_idx;
      int parent_tb_idx = selected_index[bid * (draft_token_num - 1) + i - 1] / topk;
      int parent_position = 0;
      if (parent_tb_idx > 0) {
        int parent_token_idx = parent_list[bid * (topk * (depth - 1) + 1) + parent_tb_idx];
        for (; parent_position < draft_token_num; ++parent_position) {
          if (selected_index[bid * (draft_token_num - 1) + parent_position] == parent_token_idx) {
            ++parent_position;
            break;
          }
        }
      }
      if (parent_position == draft_token_num) {
        printf(
            "ERROR: invalid eagle tree!!! Detected a token with no parent token selected. Check the logprob. The token "
            "will be dropped.");
        continue;
      }

      if (retrive_next_token[bid * draft_token_num + parent_position] == -1) {
        retrive_next_token[bid * draft_token_num + parent_position] = i;
      } else {
        int origin_next_token = retrive_next_token[bid * draft_token_num + parent_position];
        retrive_next_token[bid * draft_token_num + parent_position] = i;
        retrive_next_sibling[bid * draft_token_num + i] = origin_next_token;
      }
    }
    retrive_index[bid * draft_token_num] = bid * draft_token_num;
  } else {
    int cur_position = tid - 1;
    while (true) {
      position += 1;
      tree_mask[token_tree_idx + cur_position] = true;
      int parent_tb_idx = selected_index[bid * (draft_token_num - 1) + cur_position] / topk;
      if (parent_tb_idx == 0) {
        break;
      }

      int token_idx = parent_list[bid * (topk * (depth - 1) + 1) + parent_tb_idx];
      for (cur_position = 0; cur_position < draft_token_num; ++cur_position) {
        if (selected_index[bid * (draft_token_num - 1) + cur_position] == token_idx) {
          break;
        }
      }
    }
    positions[bid * draft_token_num + tid] = position + seq_len;
  }
}

void build_tree_kernel_efficient(at::Tensor parent_list, at::Tensor selected_index, at::Tensor verified_seq_len,
                                 at::Tensor tree_mask, at::Tensor positions, at::Tensor retrive_index,
                                 at::Tensor retrive_next_token, at::Tensor retrive_next_sibling, int64_t topk,
                                 int64_t depth, int64_t draft_token_num) {
  // TODO (ying) check shape
  // TODO (ying) check type
  int bs = parent_list.size(0);
  dim3 grid(bs);
  dim3 block(draft_token_num);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  build_tree_efficient<<<grid, block, 0, stream>>>(
      static_cast<int64_t*>(parent_list.data_ptr()), static_cast<int64_t*>(selected_index.data_ptr()),
      static_cast<int32_t*>(verified_seq_len.data_ptr()), static_cast<bool*>(tree_mask.data_ptr()),
      static_cast<int64_t*>(positions.data_ptr()), static_cast<int64_t*>(retrive_index.data_ptr()),
      static_cast<int64_t*>(retrive_next_token.data_ptr()), static_cast<int64_t*>(retrive_next_sibling.data_ptr()),
      int32_t(topk), int32_t(depth), int32_t(draft_token_num));
}

// parent_list [bs, topk * (depth - 1) + 1)]
// selected_index [bs, draft_token_num - 1]
// verified_seq_len [bs]
// tree_mask [draft_token*(seq_len[0]+draft_token) | draft_token*(seq_len[1]+draft_token) | ..] =
// [sum(verified_seq_len)*draft_token+bs*draft_token*draft_token] positions [bs * draft_token] retrive_index [b,
// draft_token, depth + 2]
__global__ void build_tree(int64_t* parent_list, int64_t* selected_index, int32_t* verified_seq_len, bool* tree_mask,
                           int64_t* positions, int64_t* retrive_index, int topk, int depth, int draft_token_num) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  if (tid >= draft_token_num) {
    return;
  }
  int seq_tree_idx = draft_token_num * draft_token_num * bid;
  for (int i = 0; i < bid; i++) {
    seq_tree_idx += verified_seq_len[i] * draft_token_num;
  }
  int seq_len = verified_seq_len[bid];
  int token_tree_idx = seq_tree_idx + (seq_len + draft_token_num) * tid + seq_len + 1;
  for (int i = 0; i < draft_token_num - 1; i++) {
    tree_mask[token_tree_idx + i] = false;
  }

  int position = 0;
  if (tid == 0) {
    positions[bid * draft_token_num] = seq_len;
    retrive_index[bid * draft_token_num * (depth + 2)] = bid * draft_token_num;
    return;
  }

  int depends_order[10];

  int cur_position = tid - 1;
  while (true) {
    depends_order[position] = cur_position + 1;
    position += 1;
    tree_mask[token_tree_idx + cur_position] = true;
    int parent_tb_idx = selected_index[bid * (draft_token_num - 1) + cur_position] / topk;
    if (parent_tb_idx == 0) {
      break;
    }

    int token_idx = parent_list[bid * (topk * (depth - 1) + 1) + parent_tb_idx];
    for (cur_position = 0; cur_position < draft_token_num; cur_position++) {
      if (selected_index[bid * (draft_token_num - 1) + cur_position] == token_idx) {
        break;
      }
    }
    if (cur_position == draft_token_num) {
      printf(
          "ERROR: invalid eagle tree!!! Detected a token with no parent token selected. Check the logprob. The token "
          "will be dropped.");
      break;
    }
  }
  positions[bid * draft_token_num + tid] = position + seq_len;

  int is_leaf = 0;
  for (int i = 1; i < draft_token_num; i++) {
    if (tree_mask[seq_tree_idx + i * (draft_token_num + seq_len) + seq_len + tid]) {
      is_leaf++;
    }
  }
  if (is_leaf == 1) {
    for (int i = 0; i < position; i++) {
      retrive_index[(bid * (draft_token_num) + tid) * (depth + 2) + position - i] =
          depends_order[i] + bid * draft_token_num;
    }
    retrive_index[(bid * (draft_token_num) + tid) * (depth + 2)] = bid * draft_token_num;
  }
}

void build_tree_kernel(at::Tensor parent_list, at::Tensor selected_index, at::Tensor verified_seq_len,
                       at::Tensor tree_mask, at::Tensor positions, at::Tensor retrive_index, int64_t topk,
                       int64_t depth, int64_t draft_token_num) {
  // TODO (ying) check shape
  // TODO (ying) check type
  int bs = parent_list.size(0);
  dim3 grid(bs);
  dim3 block(draft_token_num);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  build_tree<<<grid, block, 0, stream>>>(
      static_cast<int64_t*>(parent_list.data_ptr()), static_cast<int64_t*>(selected_index.data_ptr()),
      static_cast<int32_t*>(verified_seq_len.data_ptr()), static_cast<bool*>(tree_mask.data_ptr()),
      static_cast<int64_t*>(positions.data_ptr()), static_cast<int64_t*>(retrive_index.data_ptr()), int32_t(topk),
      int32_t(depth), int32_t(draft_token_num));
}
